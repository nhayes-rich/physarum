#include "hip/hip_runtime.h"
﻿#include "particle.cuh"

// Particles

__device__
void particle::initialize(int seed, hiprandState* devState, particleConfig* cfg, modelConfig* mdlCfg) {
    config = &(cfg[seed % mdlCfg->numSpecies]); // retrieve the config associated with the seed
    mdlConfig = mdlCfg;
    
    // start looking wherever
    float random;
    genRandom(&random, devState);
    heading = random * 2 * M_PI; 

    // pick starting location based on model config
    if (mdlConfig->start == CENTER) {
        // everything in center pixel
        location[0] = mdlConfig->width / 2;
        location[1] = mdlConfig->height / 2;
    }
    else if (mdlConfig->start == CIRCLE) {
        // randomly distributed around ellipse width/3, height/3
        float randomCircle;
        genRandom(&randomCircle, devState);
        float rx = mdlConfig->width / 3;
        float ry = mdlConfig->height / 3;

        location[0] = cos(randomCircle * 2 * M_PI) * rx + mdlConfig->width / 2;
        location[1] = sin(randomCircle * 2 * M_PI) * ry + mdlConfig->height / 2;

        heading = randomCircle * 2 * M_PI + M_PI_2;
    }

    if (mdlConfig->numSpecies == 1) {
        // one species
        speciesMask[0] = 1;
        speciesMask[1] = 1;
        speciesMask[2] = 1;
    }
    else {
        // 3 species
        speciesMask[0] = config->index == 0;
        speciesMask[1] = config->index == 1;
        speciesMask[2] = config->index == 2;
    }

}

// a particle looks in front of it a distance of config->SO, a width of config->SW,
// based on the heading and angleOffset
// it senses in a square 
__device__
float particle::sense(float angleOffset, trailMap tm, hiprandState* devState) {
    float angle = heading + angleOffset;
    float dir[2] = { config->SO * cos(angle), config->SO * sin(angle) };
    float center[2];

    // define where our sensor is in front of the particle
    for (int i = 0; i < 2; i++) {
        center[i] = location[i] + dir[i];
    }

    float mask[3];

    // i.e. for a speciesMask = [1, 0, 0]
    // we would get mask = [2, -1, -1]
    // so particles of the same species are twice as "good"
    // particles of other species are twice as negative
    for (int k = 0; k < 3; k++) {
        mask[k] = 2 * speciesMask[k] - 1;
    }

    float sum = 0;
    for (int offsetX = -config->SW; offsetX <= config->SW; offsetX++) {
        for (int offsetY = -config->SW; offsetY <= config->SW; offsetY++) {
            int pos[2];
            pos[0] = center[0] + offsetX, pos[1] = center[1] + offsetY;
            
            // make sure new sensing position is within the bounds
            if (pos[0] >= 0 && pos[0] < mdlConfig->width && pos[1] >= 0 && pos[1] < mdlConfig->height) {

                for (int k = 0; k < 3; k++) {
                    sum += tm.get(pos[0], pos[1], k, mdlConfig) * mask[k];
                }
            }
        }
    }
    return sum;
}

__device__
void particle::move(trailMap tm, hiprandState* devState) {
    float cosHeading = cos(heading);
    float sinHeading = sin(heading);

    float newX = config->SS * cosHeading + location[0];
    float newY = config->SS * sinHeading + location[1];

    int indX = (int)floor(newX);
    int indY = (int)floor(newY);

    // pick a new direction until we pick one where our move
    // puts us in a valid position
    while (indX < 0 || indX >= mdlConfig->width || indY < 0 || indY >= mdlConfig->height) {
        float random;
        genRandom(&random, devState);
        heading += random * 2 * M_PI;

        cosHeading = cos(heading);
        sinHeading = sin(heading);

        newX = config->SS * cosHeading + location[0];
        newY = config->SS * sinHeading + location[1];

        indX = (int)floor(newX);
        indY = (int)floor(newY);
    }

    location[0] = newX, location[1] = newY;

    // special case for a uniform deposit
    if (config->index == -1) {
        tm.set(config->depositT, indX, indY, 0, mdlConfig);
        tm.set(config->depositT, indX, indY, 1, mdlConfig);
        tm.set(config->depositT, indX, indY, 2, mdlConfig);
    }                         
    else {                    
        tm.set(config->depositT, indX, indY, config->index, mdlConfig);
    }


}

__device__
void particle::update(trailMap tm, hiprandState* devState) {
    move(tm, devState);

    // sense in 3 directions
    float f = sense(0, tm, devState);
    float fl = sense(config->SA, tm, devState);
    float fr = sense(-config->SA, tm, devState);

    float random;
    genRandom(&random, devState);

    if (f > fr && f > fl) {
        // if we sensed more good in front
        // than to either side, don't change
    }
    else if (f < fr && f < fl) {
        // if we sensed less forward than left and less forward than right
        // then pick either direction randomly
        heading += (2 * (random - .5)) * config->RA;
    }
    else if (fl < fr) {
        // if left was less than right
        // go right
        heading -= random * config->RA;
    }
    else if (fr < fl) {
        // if right was less than left
        // go left
        heading += random * config->RA;
    }
}
